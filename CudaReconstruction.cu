#include "hip/hip_runtime.h"
#ifndef _CudaReconstruction_
#define _CudaReconstruction_

#include <stdio.h>
#include <vector>
#include "vtkPointData.h"
#include "vtkDoubleArray.h"
#include "vtkMatrix4x4.h"
#include <math.h>
#include "ReconstructionData.h"

// ----------------------------------------------------------------------------
// Define texture and constants
__constant__ double c_gridMatrix[16];
__constant__ double3 c_gridOrig;
__constant__ int3 c_gridDims;
__constant__ double3 c_gridSpacing;
__constant__ int2 c_depthMapDims;

// ----------------------------------------------------------------------------
#define CudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}


// ----------------------------------------------------------------------------
__device__ void transformFrom4Matrix(double matrix[16], double point[3], double output[3])
{
  output[0] = matrix[0] * point[0] + matrix[1] * point[1] + matrix[2] * point[2] + matrix[3];
  output[1] = matrix[4] * point[0] + matrix[5] * point[1] + matrix[6] * point[2] + matrix[7];
  output[2] = matrix[8] * point[0] + matrix[9] * point[1] + matrix[10] * point[2] + matrix[11];
}


// ----------------------------------------------------------------------------
__device__ double norm(double vec[3])
{
  return sqrt(vec[0] * vec[0] + vec[1] * vec[1] + vec[2] * vec[2]);
}

// ----------------------------------------------------------------------------
__device__ double cumulFunction(double diff, double currentVal)
{
  double shift = 10 - 0.5 * std::abs(diff);
  if (shift < 0)
    shift = 0;
  return currentVal + shift;
}

// ----------------------------------------------------------------------------
__device__ int computeVoxelID(int coordinates[3], int type)
{
  int dimX = c_gridDims.x - 1;
  int dimY = c_gridDims.y - 1;
  if (type == 0)
    {
    dimX = c_depthMapDims.x;
    dimY = c_depthMapDims.y;
    }
  int i = coordinates[0];
  int j = coordinates[1];
  int k = coordinates[2];
  return (k*dimY + j)*dimX + i;
}

__device__ void computeVoxelCenter(int voxelCoordinate[3], double output[3])
{
  output[0] = c_gridOrig.x + (voxelCoordinate[0] + 0.5) * c_gridSpacing.x;
  output[1] = c_gridOrig.y + (voxelCoordinate[1] + 0.5) * c_gridSpacing.y;
  output[2] = c_gridOrig.z + (voxelCoordinate[2] + 0.5) * c_gridSpacing.z;
}


// ----------------------------------------------------------------------------
__global__ void depthMapKernel(double* depths, double matrixK[16], double matrixTR[16],
  double* output)
{
  // Get voxel coordinate according to thread id
  int i = threadIdx.x;
  int j = blockIdx.y;
  int k = blockIdx.z;
  int voxelCoordinate[3] = { i, j, k };

  // Get the center of the voxel
  double voxelCenterTemp[3];
  computeVoxelCenter(voxelCoordinate, voxelCenterTemp);

  // Transform voxel from grid to real coord
  double voxelCenter[3];
  transformFrom4Matrix(c_gridMatrix, voxelCenterTemp, voxelCenter);

  // Transform voxel center from real coord to camera coords
  double voxelCenterCamera[3];
  transformFrom4Matrix(matrixTR, voxelCenter, voxelCenterCamera);

  // Transform voxel center from camera coords to depth map homogeneous coords
  double voxelCenterHomogen[3];
  transformFrom4Matrix(matrixK, voxelCenterCamera, voxelCenterHomogen);

  // Get voxel center on depth map coord
  double voxelCenterDepthMap[2];
  voxelCenterDepthMap[0] = voxelCenterHomogen[0] / voxelCenterHomogen[2];
  voxelCenterDepthMap[1] = voxelCenterHomogen[1] / voxelCenterHomogen[2];

  // Get real pixel position (approximation)
  int pixel[3];
  pixel[0] = round(voxelCenterDepthMap[0]);
  pixel[1] = round(voxelCenterDepthMap[1]);
  pixel[2] = 0;

  // Get the distance between voxel and camera
  double realDepth = norm(voxelCenterCamera);

  // Test if coordinate are inside depth map
  if (pixel[0] < 0 || pixel[1] < 0 || pixel[2] < 0 ||
    pixel[0] > c_depthMapDims.x - 1 ||
    pixel[1] > c_depthMapDims.y - 1)
    {
      return;
    }

  // Compute the ID on depthmap values according to pixel position and dpeth map dimensions
  int depthMapId = computeVoxelID(pixel, 0);
  int gridId = computeVoxelID(voxelCoordinate, 1);
  double depth = depths[depthMapId];
  double currentScalarValue = output[gridId];
  double newValue = cumulFunction(realDepth - depth, currentScalarValue);

  // Update the value to the output
  output[gridId] = newValue;
}


// ----------------------------------------------------------------------------
__host__ void vtkMatrixToDoubleTable(vtkMatrix4x4* matrix, double* output)
{
  output[0] = matrix->GetElement(0, 0);
  output[1] = matrix->GetElement(0, 1);
  output[2] = matrix->GetElement(0, 2);
  output[3] = matrix->GetElement(0, 3);
  output[4] = matrix->GetElement(1, 0);
  output[5] = matrix->GetElement(1, 1);
  output[6] = matrix->GetElement(1, 2);
  output[7] = matrix->GetElement(1, 3);
  output[8] = matrix->GetElement(2, 0);
  output[9] = matrix->GetElement(2, 1);
  output[10] = matrix->GetElement(2, 2);
  output[11] = matrix->GetElement(2, 3);
  output[12] = matrix->GetElement(3, 0);
  output[13] = matrix->GetElement(3, 1);
  output[14] = matrix->GetElement(3, 2);
  output[15] = matrix->GetElement(3, 3);
}


// ----------------------------------------------------------------------------
__host__ void vtkDoubleArrayToTable(vtkDoubleArray* doubleArray, double* output)
{
  for (int i = 0; i < doubleArray->GetNumberOfTuples(); i++)
  {
    output[i] = doubleArray->GetTuple1(i);
  }
}


// ----------------------------------------------------------------------------
__host__ void vtkImageDataToTable(vtkImageData* image, double* output)
{
  vtkDoubleArray* depths = vtkDoubleArray::SafeDownCast(image->GetPointData()->GetArray("Depths"));
  vtkDoubleArrayToTable(depths, output);
}


// ----------------------------------------------------------------------------
__host__ void doubleTableToVtkDoubleArray(double* table, vtkDoubleArray* output)
{
  int nbVoxels = output->GetNumberOfTuples();
  for (int i = 0; i < nbVoxels; i++)
  {
    output->SetTuple1(i, table[i]);
  }
}


// ----------------------------------------------------------------------------
/** Main function **/
int reconstruction(std::vector<ReconstructionData*> h_dataList, // List of depth matrix and associated matrix
                   vtkMatrix4x4* i_gridMatrix, // Matrix to transform grid voxel to real coordinates
                   int h_gridDims[3], // Dimensions of the output volume
                   double h_gridOrig[3], // Origin of the output volume
                   double h_gridSpacing[3], // Spacing of the output volume
                   vtkDoubleArray* io_outScalar)
{
  if (h_dataList.size() == 0)
    return -1;

  // Get usefull value for allocation of variables
  const int matrix4Size = 16;
  const int nbPixelOnDepthMap = h_dataList[0]->GetDepthMap()->GetNumberOfPoints();
  const int nbVoxels = io_outScalar->GetNumberOfTuples();

  // Fill texture and constant values
  double* h_gridMatrix = new double[16];
  vtkMatrixToDoubleTable(i_gridMatrix, h_gridMatrix);
  double* h_outScalar = new double[nbVoxels];
  vtkDoubleArrayToTable(io_outScalar, h_outScalar);


  // Create and fill device value
  hipMemcpyToSymbol(HIP_SYMBOL(c_gridMatrix), h_gridMatrix, 16 * sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(c_gridDims), h_gridDims, 3 * sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(c_gridOrig), h_gridOrig, 3 * sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(c_gridSpacing), h_gridSpacing, 3 * sizeof(double));
  double* d_outScalar;
  CudaErrorCheck(hipMalloc((void**)&d_outScalar, nbVoxels * sizeof(double)));
  CudaErrorCheck(hipMemcpy(d_outScalar, h_outScalar, nbVoxels * sizeof(double), hipMemcpyHostToDevice));


  int h_dimDepthMap[3];
  h_dataList[0]->GetDepthMap()->GetDimensions(h_dimDepthMap);
  CudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_depthMapDims), h_dimDepthMap, 2 * sizeof(int)));

  // Organize threads into blocks and grids
  dim3 dimBlock(h_gridDims[0] - 1, 1, 1); // nb threads
  dim3 dimGrid(1, h_gridDims[1] - 1, h_gridDims[2] - 1); // nb blocks

  // Create device data from host value
  double *d_depthMap, *d_matrixK, *d_matrixRT;
  CudaErrorCheck(hipMalloc((void**)&d_depthMap, nbPixelOnDepthMap * sizeof(double)));
  CudaErrorCheck(hipMalloc((void**)&d_matrixK, matrix4Size * sizeof(double)));
  CudaErrorCheck(hipMalloc((void**)&d_matrixRT, matrix4Size * sizeof(double)));
  
  for (int i = 0; i < h_dataList.size(); i++)
    {
    // Get data and transform its properties to atomic type
    ReconstructionData* currentData = h_dataList[i];
    double* h_depthMap = new double[nbPixelOnDepthMap];
    vtkImageDataToTable(currentData->GetDepthMap(), h_depthMap);
    double* h_matrixK = new double[16];
    vtkMatrixToDoubleTable(currentData->Get4MatrixK(), h_matrixK);
    double* h_matrixRT = new double[16];
    vtkMatrixToDoubleTable(currentData->GetMatrixTR(), h_matrixRT);

    CudaErrorCheck(hipMemcpy(d_depthMap, h_depthMap, nbPixelOnDepthMap * sizeof(double), hipMemcpyHostToDevice));
    CudaErrorCheck(hipMemcpy(d_matrixK, h_matrixK, matrix4Size * sizeof(double), hipMemcpyHostToDevice));
    CudaErrorCheck(hipMemcpy(d_matrixRT, h_matrixRT, matrix4Size * sizeof(double), hipMemcpyHostToDevice));

    // run code into device
    depthMapKernel << <dimGrid, dimBlock >> >(d_depthMap, d_matrixK, d_matrixRT, d_outScalar);

    // Wait that all threads have finished
    CudaErrorCheck(hipDeviceSynchronize());

    // clean code
    delete(h_depthMap);
    delete(h_matrixK);
    delete(h_matrixRT);
    }

  // Transfer data from device to host
  hipMemcpy(h_outScalar, d_outScalar, nbVoxels * sizeof(double), hipMemcpyDeviceToHost);

  // Transfer host data to output
  doubleTableToVtkDoubleArray(h_outScalar, io_outScalar);

  // Clean memory
  hipFree(d_outScalar);
  delete(h_gridMatrix);
  delete(h_outScalar);
  hipFree(d_depthMap);
  hipFree(d_matrixK);
  hipFree(d_matrixRT);

  return 1;
}





// --------------------------------TEST----------------------------------------
// ----------------------------------------------------------------------------
__global__ void kernel(int *a, int *b)
{
  a[threadIdx.x] = b[threadIdx.x];
}
// ----------------------------------------------------------------------------
int cuda_reconstruction(
  double h_gridMatrix[16], double h_gridOrig[3], int h_gridDims[3], double h_gridSpacing[3],
  int h_depthMapDims[3], double* h_depths, double h_depthMapMatrixK[16], double h_depthMapMatrixTR[16],
  double* h_outScalar)
{
  const int N = 5;

  // create data into host
  int h_a[N] = { 0, 0, 0, 0, 0 };
  int h_b[N] = { 1, 2, 4, 5, 2 };

  // tranfer data from host to device
  int *d_a, *d_b;
  hipMalloc((void**)&d_a, N * sizeof(int));
  hipMalloc((void**)&d_b, N * sizeof(int));
  CudaErrorCheck(hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice));
  CudaErrorCheck(hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice));

  // organize threads into blocks and grids
  dim3 dimBlock(N, 1, 1); // nb threads per blocks
  dim3 dimGrid(1, 1, 1); // nb blocks

  // run code into device
  kernel << <dimGrid, dimBlock >> >(d_a, d_b);

  // transfer data from device to host
  CudaErrorCheck(hipMemcpy(h_a, d_a, N * sizeof(int), hipMemcpyDeviceToHost));
  for (int i = 0; i < N; i++)
  {
    h_outScalar[i] = h_a[i];
  }

  // free memory
  hipFree(d_a);
  hipFree(d_b);

  return 1;
}


#endif
