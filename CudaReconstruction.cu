#ifndef _CudaReconstruction_
#define _CudaReconstruction_

__global__ void kernel(int *a, int *b)
{
  a[threadIdx.x] = b[threadIdx.x];
}

int cuda_reconstruction(
    double h_gridMatrix[16], double h_gridOrig[3], int h_gridDims[3], double h_gridSpacing[3],
    int h_depthMapDims[3], double* h_depths, double h_depthMapMatrixK[16], double h_depthMapMatrixTR[16],
    double* h_outScalar)
{
  const int N = 5;

  // create data into host
  int h_a[N] = { 0, 0, 0, 0, 0 };
  int h_b[N] = { 1, 1, 1, 1, 1 };

  // tranfer data from host to device
  int *d_a, *d_b;
  hipMalloc((void**)&d_a, N * sizeof(int));
  hipMalloc((void**)&d_b, N * sizeof(int));
  hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);

  // organize threads into blocks and grids
  dim3 dimBlock(N, 1, 1);
  dim3 dimGrid(1, 1, 1);

  // run code into device
  kernel<<<dimGrid, dimBlock>>>(d_a, d_b);

  // transfer data from device to host
  hipMemcpy(h_a, d_a, N * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++)
  {
    h_outScalar[i] = h_a[i];
  }

  // free memory
  hipFree(d_a);
  hipFree(d_b);

  return 1;
}

#endif
