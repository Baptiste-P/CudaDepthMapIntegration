#include "hip/hip_runtime.h"
// Copyright(c) 2016, Kitware SAS
// www.kitware.fr
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met :
//
// 1. Redistributions of source code must retain the above copyright notice,
// this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright notice,
// this list of conditions and the following disclaimer in the documentation and
// / or other materials provided with the distribution.
//
// 3. Neither the name of the copyright holder nor the names of its contributors
// may be used to endorse or promote products derived from this software without
// specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED.IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE
// GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
// LIABILITY, OR TORT(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT
// OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#ifndef _CudaReconstruction_
#define _CudaReconstruction_

// Project include
#include "ReconstructionData.h"

// STD include
#include <math.h>
#include <stdio.h>
#include <vector>
#include <string>

// VTK includes
#include "vtkPointData.h"
#include "vtkDoubleArray.h"
#include "vtkMatrix4x4.h"
#include "vtkImageData.h"
#include "vtkNew.h"
#include "vtkCellData.h"
#include "vtkXMLImageDataWriter.h"

#define SizeMat4x4 16
#define SizePoint3D 3
#define SizeDim3D 3
// Apply to matrix, computes on 3D point
typedef double TypeCompute;

// ----------------------------------------------------------------------------
/* Define texture and constants */
__constant__ TypeCompute c_gridMatrix[SizeMat4x4]; // Matrix to transpose from basic axis to output volume axis
__constant__ TypeCompute c_gridOrig[SizePoint3D]; // Origin of the output volume
__constant__ int3 c_gridDims; // Dimensions of the output volume
__constant__ TypeCompute c_gridSpacing[SizeDim3D]; // Spacing of the output volume
__constant__ int2 c_depthMapDims; // Dimensions of all depths map
__constant__ int3 c_tileDims; // Dimensions of the tiles
__constant__ int c_nbVoxels; // Total number of voxels
__constant__ TypeCompute c_rayPotentialThick; // Thickness threshold for the ray potential function
__constant__ TypeCompute c_rayPotentialRho; // Rho at the Y axis for the ray potential function
__constant__ TypeCompute c_rayPotentialEta;
__constant__ TypeCompute c_rayPotentialDelta;
double ch_gridOrigin[3];
int ch_gridDims[3];
double ch_gridSpacing[3];
int h_tileDims[3];

// ----------------------------------------------------------------------------
/* Macro called to catch cuda error when cuda functions is called */
#define CudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

__device__ void computeVoxelCenter(int voxelCoordinate[SizePoint3D], TypeCompute output[SizePoint3D])
{
output[0] = c_gridOrig[0] + (voxelCoordinate[0] + 0.5) * c_gridSpacing[0];
output[1] = c_gridOrig[1] + (voxelCoordinate[1] + 0.5) * c_gridSpacing[1];
output[2] = c_gridOrig[2] + (voxelCoordinate[2] + 0.5) * c_gridSpacing[2];
}


// ----------------------------------------------------------------------------
/* Apply a 4x4 matrix to a 3D points */
__device__ void transformFrom4Matrix(TypeCompute M[SizeMat4x4], TypeCompute point[SizePoint3D], TypeCompute output[SizePoint3D])
{
  output[0] = M[0 * 4 + 0] * point[0] + M[0 * 4 + 1] * point[1] + M[0 * 4 + 2] * point[2] + M[0 * 4 + 3];
  output[1] = M[1 * 4 + 0] * point[0] + M[1 * 4 + 1] * point[1] + M[1 * 4 + 2] * point[2] + M[1 * 4 + 3];
  output[2] = M[2 * 4 + 0] * point[0] + M[2 * 4 + 1] * point[1] + M[2 * 4 + 2] * point[2] + M[2 * 4 + 3];
}


// ----------------------------------------------------------------------------
/* Compute the norm of a table with 3 double */
__device__ TypeCompute norm(TypeCompute vec[SizeDim3D])
{
  return sqrt(vec[0] * vec[0] + vec[1] * vec[1] + vec[2] * vec[2]);
}

// ----------------------------------------------------------------------------
/* Ray potential function which computes the increment to the current voxel */
template<typename TVolumetric>
__device__ void rayPotential(TypeCompute realDistance, TypeCompute depthMapDistance, TVolumetric& res)
{
  TypeCompute diff = (realDistance - depthMapDistance);

  TypeCompute absoluteDiff = abs(diff);
  // Can't divide by zero
  int sign = diff != 0 ? diff / absoluteDiff : 0;

  if (absoluteDiff > c_rayPotentialDelta)
    res = diff > 0 ? 0 : - c_rayPotentialEta * c_rayPotentialRho;
  else if (absoluteDiff > c_rayPotentialThick)
    res = c_rayPotentialRho * sign;
  else
    res = (c_rayPotentialRho / c_rayPotentialThick) * diff;
}

// ----------------------------------------------------------------------------
/* Compute the voxel Id on a 1D table according to its 3D coordinates
  coordinates : 3D coordinates
*/
int computeVoxelIDGrid(int coordinates[SizePoint3D])
{
  int dimX = ch_gridDims[0] - 1;
  int dimY = ch_gridDims[1] - 1;
  int i = coordinates[0];
  int j = coordinates[1];
  int k = coordinates[2];
  return (k*dimY + j)*dimX + i;
}

// ----------------------------------------------------------------------------
/* Compute the voxel's 3D coordinates in tile according to its ID on a 1D table
  gridID : 1D coordinate
  coordinates : 3D coordinates
*/
void computeVoxel3DCoords(int gridId, int tileDims[3], int coordinates[SizePoint3D])
{
  coordinates[0] = gridId % tileDims[0];
  coordinates[1] = (gridId / tileDims[0]) % tileDims[1];
  coordinates[2] = ((gridId / tileDims[0]) / tileDims[1]) % tileDims[2];
}

// ----------------------------------------------------------------------------
/* Compute the tiles' origins as 3D coordinates according to their size and
 * the size of the voxel grid
  nbTilesXYZ : number of tiles in each dimension
  tileOrigin : 3D coordinates
*/
void computeTileOrigins(int nbTilesXYZ[3], int tileOrigin[][3])
{
  for (int x = 0; x < nbTilesXYZ[0]; x++)
  {
    for (int y = 0; y < nbTilesXYZ[1]; y++)
    {
      for (int z = 0; z < nbTilesXYZ[2]; z++)
      {
        int id = z + nbTilesXYZ[2]*(y + nbTilesXYZ[1]*x);
        tileOrigin[id][0] = x * h_tileDims[0];
        tileOrigin[id][1] = y * h_tileDims[1];
        tileOrigin[id][2] = z * h_tileDims[2];

        std::cout<<"tileOrigin["<<id<<"] : "<<tileOrigin[id][0]<<" "<<tileOrigin[id][1]<<" "<<tileOrigin[id][2]<<std::endl;
      }
    }
  }
}

// ----------------------------------------------------------------------------
/* Compute the tiles' dimensions to use all GPUs
*/
template<typename TVolumetric>
void computeTileDims(int nbDevices)
{
  // Initialize tile sizes such as there is one tile per device
  h_tileDims[0] = ch_gridDims[0] - 1;
  h_tileDims[1] = ch_gridDims[1] - 1;
  h_tileDims[2] = vtkMath::Ceil(double(ch_gridDims[2] - 1)) / nbDevices;

  size_t freeMemory, free, totalMemory;

  // Find the minimum amount of free memory on the devices
  for (int i = 0; i < nbDevices; i++)
  {
    CudaErrorCheck(hipSetDevice(i));
    CudaErrorCheck(hipMemGetInfo(&free, &totalMemory));

    if (i == 0)
    {
      freeMemory = free;
    }
    else
    {
      freeMemory = std::min(freeMemory, free);
    }
  }

  int voxelsPerTile = h_tileDims[0] * h_tileDims[1] * h_tileDims[2];
  int usagePercent = 80;
  int freeVoxels = double(usagePercent * freeMemory) / (100 * sizeof(TVolumetric));

  // Use free GPU memory to reduce tile sizes if need be
  while (voxelsPerTile > freeVoxels)
  {
    // Subdivide the Z dimension
    if (h_tileDims[2] > 1)
    {
      h_tileDims[2] = vtkMath::Ceil(double(h_tileDims[2]) / 2);
    }
    else
    {
      // Subdivide the Y dimension
      if (h_tileDims[1] > 1)
      {
        h_tileDims[1] = vtkMath::Ceil(double(h_tileDims[1]) / 2);
      }
      else
      {
        // Subdivide the X dimension
        h_tileDims[0] = vtkMath::Ceil(double(h_tileDims[0]) / 2);
      }
    }

    voxelsPerTile = h_tileDims[0] * h_tileDims[1] * h_tileDims[2];
  }
}

// ----------------------------------------------------------------------------
/* Copy the tile data to its spatial region in output scalar
  tileOrigin : 3D coordinates
*/
template<typename TVolumetric>
void copyTileDataToOutput(int nbVoxelsTile, int tileId, int tileOrigin[3], TVolumetric* outTile, TVolumetric* outScalar)
{
  for (int k = 0; k < nbVoxelsTile; k++)
  {
    int voxelIndexRelative[SizePoint3D];
    computeVoxel3DCoords(k, h_tileDims, voxelIndexRelative);

    int voxelIndex[SizePoint3D];
    voxelIndex[0] = tileOrigin[0] + voxelIndexRelative[0];
    voxelIndex[1] = tileOrigin[1] + voxelIndexRelative[1];
    voxelIndex[2] = tileOrigin[2] + voxelIndexRelative[2];

    if (voxelIndex[0] < ch_gridDims[0] - 1
    && voxelIndex[1] < ch_gridDims[1] - 1
    && voxelIndex[2] < ch_gridDims[2] - 1)
    {
      int gridId = computeVoxelIDGrid(voxelIndex);
      outScalar[gridId] = outTile[k];
    }
  }
}

// ----------------------------------------------------------------------------
/* Compute the voxel relative Id on a 1D table according to its 3D coordinates
  coordinates : 3D coordinates
*/
__device__ int computeVoxelRelativeIDGrid(int coordinates[SizePoint3D])
{
  int dimX = c_tileDims.x;
  int dimY = c_tileDims.y;
  int i = coordinates[0];
  int j = coordinates[1];
  int k = coordinates[2];
  return (k*dimY + j)*dimX + i;
}


// ----------------------------------------------------------------------------
/* Compute the pixel Id on a 1D table according to its 3D coordinates
  (third coordinate is not used)
coordinates : 3D coordinates
*/
__device__ int computeVoxelIDDepth(int coordinates[SizePoint3D])
{
  int dimX = c_depthMapDims.x;
  int dimY = c_depthMapDims.y;
  int x = coordinates[0];
  int y = coordinates[1];
  // /!\ vtkImageData has its origin at the bottom left, not top left
  return (dimX*(dimY-1-y)) + x;
}

// ----------------------------------------------------------------------------
/* Main function called inside the kernel
  depths : depth map values
  matrixK : matrixK
  matrixTR : matrixTR
  output : double table that will be filled at the end of function
*/
template<typename TVolumetric>
__global__ void depthMapKernel(int d_tileOrigin[3], TypeCompute* depths, TypeCompute matrixK[SizeMat4x4],
  TypeCompute matrixTR[SizeMat4x4], TVolumetric* output)
{
  // Get relative voxel coordinate of the voxel according to thread id
  int voxelIndexRelative[SizePoint3D] = { (int)threadIdx.x, (int)blockIdx.y, (int)blockIdx.z };
  int gridIdRelative = computeVoxelRelativeIDGrid(voxelIndexRelative);

  // Get true voxel coordinate
  int voxelIndex[SizePoint3D];
  voxelIndex[0] = d_tileOrigin[0] + voxelIndexRelative[0];
  voxelIndex[1] = d_tileOrigin[1] + voxelIndexRelative[1];
  voxelIndex[2] = d_tileOrigin[2] + voxelIndexRelative[2];

  // Don't process out of bounds voxels
  if (voxelIndex[0] < c_gridDims.x - 1
  && voxelIndex[1] < c_gridDims.y - 1
  && voxelIndex[2] < c_gridDims.z - 1)
  {
    TypeCompute voxelCenterCoordinate[SizePoint3D];
    computeVoxelCenter(voxelIndex, voxelCenterCoordinate);
    TypeCompute voxelCenter[SizePoint3D];
    transformFrom4Matrix(c_gridMatrix, voxelCenterCoordinate, voxelCenter);

    // Transform voxel center from real coord to camera coords
    TypeCompute voxelCenterCamera[SizePoint3D];
    transformFrom4Matrix(matrixTR, voxelCenter, voxelCenterCamera);

    // Transform voxel center from camera coords to depth map homogeneous coords
    TypeCompute voxelCenterHomogen[SizePoint3D];
    transformFrom4Matrix(matrixK, voxelCenterCamera, voxelCenterHomogen);
    if (voxelCenterHomogen[2] < 0)
      {
      return;
      }
    // Get voxel center on depth map coord
    TypeCompute voxelCenterDepthMap[2];
    voxelCenterDepthMap[0] = voxelCenterHomogen[0] / voxelCenterHomogen[2];
    voxelCenterDepthMap[1] = voxelCenterHomogen[1] / voxelCenterHomogen[2];
    // Get real pixel position (approximation)
    int pixel[SizePoint3D];
    pixel[0] = round(voxelCenterDepthMap[0]);
    pixel[1] = round(voxelCenterDepthMap[1]);
    pixel[2] = 0;

    // Test if coordinate are inside depth map
    if (pixel[0] < 0 || pixel[1] < 0 ||
      pixel[0] >= c_depthMapDims.x ||
      pixel[1] >= c_depthMapDims.y )
      {
      return;
      }

    // Compute the ID on depthmap values according to pixel position and depth map dimensions
    int depthMapId = computeVoxelIDDepth(pixel);
    TypeCompute depth = depths[depthMapId];
    if (depth == -1)
      {
      return;
      }

    TypeCompute realDepth = voxelCenterCamera[2];
    TVolumetric newValue;
    rayPotential<TVolumetric>(realDepth, depth, newValue);
    // Update the value to the output
    output[gridIdRelative] += newValue;
  }
}





// ----------------------------------------------------------------------------
/* Extract data from a 4x4 vtkMatrix and fill a double table with 16 space */
__host__ void vtkMatrixToTypeComputeTable(vtkMatrix4x4* matrix, TypeCompute* output)
{
  int cpt = 0;
  for (int i = 0; i < 4; i++)
    {
    for (int j = 0; j < 4; j++)
      {
      output[cpt++] = (TypeCompute)matrix->GetElement(i, j);
      }
    }
}


// ----------------------------------------------------------------------------
/* Extract double value from vtkDoubleArray and fill a double table (output) */
template <typename T>
__host__ void vtkDoubleArrayToTable(vtkDoubleArray* doubleArray, T* output)
{
  for (int i = 0; i < doubleArray->GetNumberOfTuples(); i++)
  {
    output[i] = (T)doubleArray->GetTuple1(i);
  }
}


// ----------------------------------------------------------------------------
/* Extract point data array (name 'Depths') from vtkImageData and fill a double table */
__host__ void vtkImageDataToTable(vtkImageData* image, TypeCompute* output)
{
  vtkDoubleArray* depths = vtkDoubleArray::SafeDownCast(image->GetPointData()->GetArray("Depths"));
  vtkDoubleArrayToTable<TypeCompute>(depths, output);
}


// ----------------------------------------------------------------------------
/* Fill a vtkDoubleArray from a double table */
template<typename TVolumetric>
__host__ void doubleTableToVtkDoubleArray(TVolumetric* table, vtkDoubleArray* output)
{
  int nbVoxels = output->GetNumberOfTuples();
  for (int i = 0; i < nbVoxels; i++)
  {
    output->SetTuple1(i, (double)table[i]);
  }
}


// ----------------------------------------------------------------------------
/* Initialize cuda constant */
void CudaInitialize(vtkMatrix4x4* i_gridMatrix, // Matrix to transform grid voxel to real coordinates
                int h_gridDims[SizeDim3D], // Dimensions of the output volume
                double h_gridOrig[SizePoint3D], // Origin of the output volume
                double h_gridSpacing[SizeDim3D], // Spacing of the output volume
                double h_rayPThick,
                double h_rayPRho,
                double h_rayPEta,
                double h_rayPDelta,
                int h_depthMapDims[2],
                int h_tilingDims[SizeDim3D])
{

  TypeCompute* h_gridMatrix = new TypeCompute[SizeMat4x4];
  vtkMatrixToTypeComputeTable(i_gridMatrix, h_gridMatrix);

  hipMemcpyToSymbol(HIP_SYMBOL(c_gridMatrix), h_gridMatrix, SizeMat4x4 * sizeof(TypeCompute));
  hipMemcpyToSymbol(HIP_SYMBOL(c_gridDims), h_gridDims, SizeDim3D * sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(c_gridOrig), h_gridOrig, SizePoint3D * sizeof(TypeCompute));
  hipMemcpyToSymbol(HIP_SYMBOL(c_gridSpacing), h_gridSpacing, SizeDim3D * sizeof(TypeCompute));
  hipMemcpyToSymbol(HIP_SYMBOL(c_rayPotentialThick), &h_rayPThick, sizeof(TypeCompute));
  hipMemcpyToSymbol(HIP_SYMBOL(c_rayPotentialRho), &h_rayPRho, sizeof(TypeCompute));
  hipMemcpyToSymbol(HIP_SYMBOL(c_rayPotentialEta), &h_rayPEta, sizeof(TypeCompute));
  hipMemcpyToSymbol(HIP_SYMBOL(c_rayPotentialDelta), &h_rayPDelta, sizeof(TypeCompute));
  hipMemcpyToSymbol(HIP_SYMBOL(c_depthMapDims), h_depthMapDims, 2 * sizeof(int));

  ch_gridOrigin[0] = h_gridOrig[0];
  ch_gridOrigin[1] = h_gridOrig[1];
  ch_gridOrigin[2] = h_gridOrig[2];

  ch_gridDims[0] = h_gridDims[0];
  ch_gridDims[1] = h_gridDims[1];
  ch_gridDims[2] = h_gridDims[2];

  ch_gridSpacing[0] = h_gridSpacing[0];
  ch_gridSpacing[1] = h_gridSpacing[1];
  ch_gridSpacing[2] = h_gridSpacing[2];

  h_tileDims[0] = h_tilingDims[0];
  h_tileDims[1] = h_tilingDims[1];
  h_tileDims[2] = h_tilingDims[2];

  // Clean memory
  delete(h_gridMatrix);

}

// ----------------------------------------------------------------------------
/* Read all depth map and process each of them. Fill the output 'io_scalar' */
template <typename TVolumetric>
bool ProcessDepthMap(std::vector<std::string> vtiList,
                     std::vector<std::string> krtdList,
                     double thresholdBestCost,
                     vtkDoubleArray* io_scalar)
{
  if (vtiList.size() == 0 || krtdList.size() == 0)
    {
    std::cerr << "Error, no depthMap or KRTD matrix have been loaded" << std::endl;
    return false;
    }

  // Define usefull constant values
  ReconstructionData* data = new ReconstructionData(vtiList[0], krtdList[0]);
  const int nbPixelOnDepthMap = data->GetDepthMap()->GetNumberOfPoints();
  const int nbVoxels = io_scalar->GetNumberOfTuples();
  CudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_nbVoxels), &nbVoxels, sizeof(int)));
  const int nbDepthMap = (int)vtiList.size();
  int nbDevices;
  hipGetDeviceCount(&nbDevices);
  std::cout << "number of devices : " << nbDevices << std::endl;

  std::cout << "START CUDA ON " << nbDepthMap << " Depth maps" << std::endl;

  // Create depthmap device data from host value
  TypeCompute *d_depthMap, *d_matrixK, *d_matrixRT;
  CudaErrorCheck(hipMalloc((void**)&d_depthMap, nbPixelOnDepthMap * sizeof(TypeCompute)));
  CudaErrorCheck(hipMalloc((void**)&d_matrixK, SizeMat4x4 * sizeof(TypeCompute)));
  CudaErrorCheck(hipMalloc((void**)&d_matrixRT, SizeMat4x4 * sizeof(TypeCompute)));
  TypeCompute* h_depthMap = new TypeCompute[nbPixelOnDepthMap];
  TypeCompute* h_matrixK = new TypeCompute[SizeMat4x4];
  TypeCompute* h_matrixRT = new TypeCompute[SizeMat4x4];

  // Runtime calculated tiling
  if (h_tileDims[0] == 0 && h_tileDims[1] == 0 && h_tileDims[2] == 0)
  {
    computeTileDims<TVolumetric>(nbDevices);
  }
  CudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(c_tileDims), h_tileDims, 3 * sizeof(int)));

  int nbTilesXYZ[3];
  // Compute the numbers of tiles needed to fill each dimension
  for (int i = 0; i < 3; i++)
  {
    nbTilesXYZ[i] = vtkMath::Ceil(double(ch_gridDims[i] - 1) / (h_tileDims[i]));
  }
  std::cout << "Tiling X : " << nbTilesXYZ[0] << ", Y : " << nbTilesXYZ[1]
  << ", Z : " << nbTilesXYZ[2] << std::endl;

  // Define tiling dimensions
  const int nbVoxelsTile = h_tileDims[0] * h_tileDims[1] * h_tileDims[2];
  const int nbTiles = nbTilesXYZ[0] * nbTilesXYZ[1] * nbTilesXYZ[2];

  // Compute and allocate tile information
  int tileOrigin[nbTiles][3];
  computeTileOrigins(nbTilesXYZ, tileOrigin);
  int* d_tileOrigin;
  CudaErrorCheck(hipMalloc((void**)&d_tileOrigin, 3 * sizeof(int)));
  TVolumetric* h_outTile = new TVolumetric[nbVoxelsTile];
  TVolumetric* d_outTile;
  CudaErrorCheck(hipMalloc((void**)&d_outTile, nbVoxelsTile * sizeof(TVolumetric)));
  CudaErrorCheck(hipMemset(d_outTile, 0, nbVoxelsTile * sizeof(TVolumetric)));

  std::cout << "Tile : " << h_tileDims[0] << "x" << h_tileDims[1] << "x" << h_tileDims[2] << std::endl;
  size_t freeMemory, totalMemory;
  CudaErrorCheck(hipMemGetInfo(&freeMemory, &totalMemory));
  std::cout << "Free nb after tiling : " << freeMemory / sizeof(TVolumetric) << std::endl << std::endl;

  // Organize threads into blocks and grids
  dim3 dimBlock(h_tileDims[0], 1, 1); // nb threads on each block
  dim3 dimGrid(1, h_tileDims[1], h_tileDims[2]); // nb blocks on a grid

  // Transform vtkDoubleArray to table
  TVolumetric* h_outScalar = new TVolumetric[nbVoxels];
  vtkDoubleArrayToTable<TVolumetric>(io_scalar, h_outScalar);

  int nbConcurrent = std::min(nbTiles, nbDevices);
  int nbSequential = vtkMath::Ceil(double(nbTiles) / nbConcurrent);

  // Tiles output init
  double gridOrigin[3];
  vtkNew<vtkImageData> gridTile;
  gridTile->SetDimensions(h_tileDims);
  gridTile->SetSpacing(ch_gridSpacing);
  gridTile->SetOrigin(0.0, 0.0, 0.0);

  vtkNew<vtkDoubleArray> outScalarTile;
  outScalarTile->SetName("reconstruction_scalar");
  outScalarTile->SetNumberOfComponents(1);
  outScalarTile->SetNumberOfTuples(gridTile->GetNumberOfCells());
  outScalarTile->FillComponent(0, 0);

  gridTile->GetCellData()->AddArray(outScalarTile.Get());

  vtkNew<vtkXMLImageDataWriter> writer;
  writer->SetInputData(gridTile.Get());
  std::string tileFileName;
  //

  // Process the tilescalar
  for (int is = 0; is < nbSequential; is++)
  {
    std::cout << "\ntile: " << (nbConcurrent * is);
    if(nbConcurrent > 1)
    {
      std::cout << "to " << std::min(nbConcurrent, nbTiles - is * nbConcurrent);
    }
    std::cout << "\t(" << (nbConcurrent * is) + std::min(nbConcurrent, nbTiles - is * nbConcurrent) << "/" << nbTiles << ")" << std::endl;

    for (int j = 0; j < nbDepthMap; j++)
    {
      if (j % (nbDepthMap / 10) == 0)
      {
        std::cout << (100 * j) / nbDepthMap <<" %\t("
        << ((100 * is) / nbSequential) + ((j * 100) / (nbDepthMap * nbSequential)) << " %)" << std::flush;
      }

      // Init depthmap data to be transfered
      if(j == 0)
      {
        ReconstructionData data(vtiList[0], krtdList[0]);
        data.ApplyDepthThresholdFilter(thresholdBestCost);

        // Get data and transform its properties to atomic type
        vtkImageDataToTable(data.GetDepthMap(), h_depthMap);
        vtkMatrixToTypeComputeTable(data.Get4MatrixK(), h_matrixK);
        vtkMatrixToTypeComputeTable(data.GetMatrixTR(), h_matrixRT);
      }

      // Copy data to devices and run kernels
      for (int ic = 0; ic < std::min(nbConcurrent, nbTiles - is * nbConcurrent); ic++)
      {
        int tileId = ic + nbConcurrent * is;

        CudaErrorCheck(hipSetDevice(ic));

        // Wait that all threads have finished on selected device
        CudaErrorCheck(hipDeviceSynchronize());

        // Copy data from host to
        CudaErrorCheck(hipMemcpy(d_tileOrigin, tileOrigin[tileId], 3 * sizeof(int), hipMemcpyHostToDevice));
        CudaErrorCheck(hipMemcpy(d_depthMap, h_depthMap, nbPixelOnDepthMap * sizeof(TypeCompute), hipMemcpyHostToDevice));
        CudaErrorCheck(hipMemcpy(d_matrixK, h_matrixK, SizeMat4x4 * sizeof(TypeCompute), hipMemcpyHostToDevice));
        CudaErrorCheck(hipMemcpy(d_matrixRT, h_matrixRT, SizeMat4x4 * sizeof(TypeCompute), hipMemcpyHostToDevice));

        // run code into device
        depthMapKernel<TVolumetric> << < dimGrid, dimBlock >> >(d_tileOrigin, d_depthMap, d_matrixK, d_matrixRT, d_outTile);
      }

      // Prepare next depthmap data while kernel is running
      if (j < nbDepthMap - 1)
      {
        ReconstructionData data(vtiList[j+1], krtdList[j+1]);
        data.ApplyDepthThresholdFilter(thresholdBestCost);

        // Get data and transform its properties to atomic type
        vtkImageDataToTable(data.GetDepthMap(), h_depthMap);
        vtkMatrixToTypeComputeTable(data.Get4MatrixK(), h_matrixK);
        vtkMatrixToTypeComputeTable(data.GetMatrixTR(), h_matrixRT);
      }
    }

    // Retrieve devices tile output and update host voxels
    for (int ic = 0; ic < std::min(nbConcurrent, nbTiles - is * nbConcurrent); ic++)
    {
      int tileId = ic + nbConcurrent * is;

      CudaErrorCheck(hipSetDevice(ic));

      // Wait that all threads have finished on selected device
      CudaErrorCheck(hipDeviceSynchronize());

      // Transfer tile data from device in host memory
      CudaErrorCheck(hipMemcpy(h_outTile, d_outTile, nbVoxelsTile * sizeof(TVolumetric), hipMemcpyDeviceToHost));

      // Reset the current device voxel tile to 0s
      CudaErrorCheck(hipMemset(d_outTile, 0, nbVoxelsTile * sizeof(TVolumetric)));

      ////
      // Convert
      doubleTableToVtkDoubleArray<TVolumetric>(h_outTile, outScalarTile.Get());
      // Save tile to vtkImageData
      // Set tile origin
      for (int k = 0; k < 3; k++)
      {
        gridOrigin[k] = ch_gridOrigin[k] + tileOrigin[tileId][k] * ch_gridSpacing[k];
      }
      gridTile->SetOrigin(gridOrigin);

      tileFileName = "tile_" + std::to_string(tileId) + ".vti";
      writer->SetFileName(tileFileName.c_str());
      writer->Write();

      // Copy data from tile to output double array
      copyTileDataToOutput<TVolumetric>(nbVoxelsTile, tileId, tileOrigin[tileId], h_outTile, h_outScalar);
    }
  }

  // Transfer host data to output
  doubleTableToVtkDoubleArray<TVolumetric>(h_outScalar, io_scalar);

  // Clean memory.
  delete(data);
  delete(h_outScalar);
  delete(h_outTile);
  hipFree(d_outTile);
  hipFree(d_depthMap);
  hipFree(d_matrixK);
  hipFree(d_matrixRT);
  delete(h_depthMap);
  delete(h_matrixK);
  delete(h_matrixRT);

  std::cout << "\r" << "100 %" << std::flush << std::endl << std::endl;

  return true;
}

// ----------------------------------------------------------------------------
// Define template for the compiler
template
bool ProcessDepthMap<float>(std::vector<std::string> vtiList,
std::vector<std::string> krtdList,
double thresholdBestCost,
vtkDoubleArray* io_scalar);

template
bool ProcessDepthMap<double>(std::vector<std::string> vtiList,
std::vector<std::string> krtdList,
double thresholdBestCost,
vtkDoubleArray* io_scalar);

#endif
